#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cmath>
#include <complex>
#include <fstream>
#include <hip/hip_runtime.h>
#include <GLUT/glut.h>
#include <cuda_gl_interop.h>
__global__
void mandelbrotGPU(float* output, int width, int height, float xmin, float xmax, float ymin, float ymax, int max_iter, float chaos_c) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float real = xmin + (xmax - xmin) * x / width;
        float imag = ymin + (ymax - ymin) * y / height;

        float2 c = make_float2(real * chaos_c, imag * chaos_c);
        float2 z = c;

        int color = max_iter;

        for (int i = 0; i < max_iter; ++i) {
            if (z.x * z.x + z.y * z.y > 4.0f) {
                color = i;
                break;
            }
            float temp = z.x;
            z.x = z.x * z.x - z.y * z.y + c.x;
            z.y = 2.0f * temp * z.y + c.y;
        }

        output[y * width + x] = static_cast<float>(color) / max_iter;
    }
}

__host__
void mandelbrotCPU(float* output, int width, int height, float xmin, float xmax, float ymin, float ymax, int max_iter, float chaos_c) {

    for (int y = 0; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            float real = xmin + (xmax - xmin) * x / (width - 1);
            float imag = ymin + (ymax - ymin) * y / (height - 1);



            float2 c = make_float2(real * chaos_c, imag * chaos_c);
            float2 z = c;

            int color = max_iter;

            for (int i = 0; i < max_iter; ++i) {
                if (z.x * z.x + z.y * z.y > 4.0f) {
                    color = i;
                    break;
                }
                float temp = z.x;
                z.x = z.x * z.x - z.y * z.y + c.x;
                z.y = 2.0f * temp * z.y + c.y;
            }

            output[y * width + x] = static_cast<float>(color) / max_iter;
        }
    }
}
__host__
void saveArrayToFile(const char* filename, float* data, int size) {
    std::ofstream file(filename, std::ios::binary);
    file.write(reinterpret_cast<char*>(data), size * sizeof(float));
    file.close();
}
__host__
void saveParameters(float xmin, float xmax, float ymin, float ymax) {
    std::ofstream file("parameters_XY.txt");
    if (file.is_open()) {
        file << xmin << " " << xmax << " " << ymin << " " << ymax;
        file.close();
        std::cout << "Parametry zapisane do pliku." << std::endl;

    }
    else {
        std::cerr << "Nie można otworzyć pliku do zapisu." << std::endl;
    }

}



__global__ void processImage(float* input, uchar4* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        float value = input[y * width + x];
        uchar4 color = make_uchar4(value * 255, value * 255, value * 255, 255);
        output[y * width + x] = color;
    }
}




int main() {

    const int width = 1600;
    const int height = 1600;
    float var_c = 0.89;

    //const float xmin = -2.0f;     const float xmax = 1.0f;
    //const float ymin = -1.5f;     const float ymax = 1.5f;

    const int max_iter = 50;

    const float xmin = -2.0f;       const float xmax = -1.54f;
    const float ymax = 0.25f;       const float ymin = -0.25f;


    saveParameters(xmin, xmax, ymin, ymax);

    float* outputCPU = new float[width * height];
    float* outputGPU;
    hipMallocManaged(&outputGPU, width * height * sizeof(float));


    //Alokacja wątków na blok
    dim3 blockSize(32, 32);
    //Alokacja bloku na siatkę
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    do
    {
        //std::cout << "Podaj zmienną c z przedziału 0-1 np. 0.45: ";
        //std::cin >> var_c;
    } while (var_c > 1.00 && var_c < 0.0);




    // Pomiar czasu dla GPU
    hipEvent_t startGPU, stopGPU;
    hipEventCreate(&startGPU);
    hipEventCreate(&stopGPU);

    hipEventRecord(startGPU);
    mandelbrotGPU << <gridSize, blockSize >> > (outputGPU, width, height, xmin, xmax, ymin, ymax, max_iter, var_c);
    hipEventRecord(stopGPU);
    hipEventSynchronize(stopGPU);

    float elapsedTimeGPU;
    hipEventElapsedTime(&elapsedTimeGPU, startGPU, stopGPU);
    std::cout << "Czas GPU: " << elapsedTimeGPU << " ms" << std::endl;


    hipEventDestroy(startGPU);
    hipEventDestroy(stopGPU);

    // Pomiar czasu dla CPU
    hipEvent_t startCPU, stopCPU;
    hipEventCreate(&startCPU);
    hipEventCreate(&stopCPU);

    hipEventRecord(startCPU);

    mandelbrotCPU(outputCPU, width, height, xmin, xmax, ymin, ymax, max_iter, var_c);
    hipEventRecord(stopCPU);
    hipEventSynchronize(stopCPU);

    float elapsedTimeCPU;
    hipEventElapsedTime(&elapsedTimeCPU, startCPU, stopCPU);
    std::cout << "Czas CPU: " << elapsedTimeCPU << " ms" << std::endl;

    hipEventDestroy(startCPU);
    hipEventDestroy(stopCPU);


    hipDeviceSynchronize();

    // Zapisz tablicę pixels do pliku
    saveArrayToFile("output_mandela_arrayGPU.bin", outputGPU, width * height);
    saveArrayToFile("output_mandela_arrayCPU.bin", outputCPU, width * height);
    hipFree(outputGPU);
    delete[] outputCPU;

    return 0;
}